#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>

#include "pcmath/pcmath.hpp"

__device__ void PrintMat2(const pcm::Mat2 &m, const char *name) {
    printf("%s = [[%f, %f],\n[%f, %f]]\n", name, m[0][0], m[1][0], m[0][1], m[1][1]);
}

__device__ void PrintMat3(const pcm::Mat3 &m, const char *name) {
    printf("%s = [[%f, %f, %f],\n[%f, %f, %f],\n[%f, %f, %f]]\n", name,
        m[0][0], m[1][0], m[2][0],
        m[0][1], m[1][1], m[2][1],
        m[0][2], m[1][2], m[2][2]
    );
}

__device__ void PrintMat4(const pcm::Mat4 &m, const char *name) {
    printf("%s = [[%f, %f, %f, %f],\n[%f, %f, %f, %f],\n[%f, %f, %f, %f],\n[%f, %f, %f, %f]]\n",
        name,
        m[0][0], m[1][0], m[2][0], m[3][0],
        m[0][1], m[1][1], m[2][1], m[3][1],
        m[0][2], m[1][2], m[2][2], m[3][2],
        m[0][3], m[1][3], m[2][3], m[3][3]
    );
}

__global__ void TestVec() {
    pcm::Vec2 v0(0.2f, 0.4f);
    pcm::Vec3 v1(v0, 3.0f);
    pcm::Vec4 v2(v1, 2.1f);

    printf("v0 = (%f, %f)\n", v0.X(), v0.Y());
    printf("v1 = (%f, %f, %f)\n", v1.X(), v1.Y(), v1.Z());
    printf("v2 = (%f, %f, %f, %f)\n", v2.X(), v2.Y(), v2.Z(), v2.W());

    pcm::Vec3 v3(0.3f, 0.5f, 1.0f);
    pcm::Vec3 v4(0.6f, 0.9f, 2.5f);
    pcm::Vec3 v5 = (v3 - v1).Cross(v4 - v1);
    printf("v5 = (%f, %f, %f)\n", v5.X(), v5.Y(), v5.Z());

    float len = (2.0f * (v4 - v3) + 1.3f * (v5 - v4)).Length();
    printf("len = %f\n", len);

    v1 = v5 + v3;
    printf("v1 (2) = (%f, %f, %f)\n", v1.X(), v1.Y(), v1.Z());

    pcm::Vec3 v6 = pcm::Vec3::UnitZ();
    printf("v6 = (%f, %f, %f)\n", v6.X(), v6.Y(), v6.Z());
}

__global__ void TestMat() {
    pcm::Mat2 m0 = pcm::Mat2::Identity();
    pcm::Mat3 m1 = pcm::Mat3::Identity() / 2.0f;
    pcm::Mat4 m2 = 2.2f * pcm::Mat4::Identity();
    PrintMat2(m0, "m0");
    PrintMat3(m1, "m1");
    PrintMat4(m2, "m2");

    pcm::Mat3 m3 = pcm::Mat3(pcm::Vec3(10.0f, 1.0f, 1.0f), pcm::Vec3(2.0f, 10.0f, 2.0f), pcm::Vec3(3.0f, 3.0f, 10.0f));
    PrintMat3(m3, "m3");

    pcm::Mat3 m4 = m3.Transpose();
    PrintMat3(m4, "m4");

    pcm::Mat3 m5 = m3.Inverse();
    pcm::Mat3 m6 = m3 * m5;
    PrintMat3(m5, "m5");
    PrintMat3(m6, "m6");
}

__global__ void TestUtils() {
    pcm::Mat4 m0 = pcm::Perspective(pcm::Radians(45.0f), 1.3f, 0.001f, 100.0f);
    PrintMat4(m0, "m0");
}

int main() {
    TestVec<<<1, 1>>>();

    TestMat<<<1, 1>>>();

    TestUtils<<<1, 1>>>();

    return 0;
}